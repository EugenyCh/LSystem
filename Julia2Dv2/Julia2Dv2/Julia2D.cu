#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "Julia2D.cuh"
#include "hipComplex.cuh"
#include <GL/freeglut.h>
#include <GL/freeglut_ext.h>
#include <stdio.h>
#include <time.h>

__global__ void kernel(byte* buffer, const int width, const int height, const float cx, const float cy, const int iters)
{
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	if (x >= width)
		return;
	int y = threadIdx.y + blockIdx.y * blockDim.y;
	if (y >= height)
		return;
	int offset = x + y * width;

	// Compute point at this position
	int halfWidth = width >> 1;
	int halfHeight = height >> 1;
	int halfSide = MAX(halfWidth, halfHeight);
	float jx = 2.0f * (float)(x - halfWidth) / halfSide;
	float jy = 2.0f * (float)(y - halfHeight) / halfSide;
	hipComplex c(cx, cy);
	hipComplex z(jx, jy);

	// Iterating
	int i;
	for (i = 0; i < iters; ++i)
	{
		z = z * z + c;
		if (z.sqrMagnitude() > 4.0f)
			break;
	}
	float k = (float)i / iters;

	// Setting point color
	buffer[offset * 3] = (byte)(k * 255);
	buffer[offset * 3 + 1] = (byte)(k * 255);
	buffer[offset * 3 + 2] = (byte)(k * 255);
}

Julia2D::Julia2D(float cx, float cy)
{
	this->cx = cx;
	this->cy = cy;
}

bool Julia2D::compute(size_t width, size_t height, int iters)
{
	if (points)
		delete[] points;
	points = new point[width * height];
	this->width = width;
	this->height = height;
	int side = MAX(width, height);

	const size_t sz = width * height;
	byte* buffer = new byte[sz * 3];
	byte* dev_buffer;

	if (hipMalloc((void**)&dev_buffer, sz * 3) != hipSuccess)
	{
		printf("Error on creating buffer of pixels in GPU\n");
		return false;
	}

	dim3 blocks((side + 31) / 32, (side + 31) / 32);
	dim3 threads(32, 32);
	printf("Rendering\n");
	clock_t tStart = clock();
	kernel<<<blocks, threads>>>(dev_buffer, width, height, cx, cy, 50);
	hipDeviceSynchronize();
	clock_t tFinish = clock();
	double tDelta = (double)(tFinish - tStart) / CLOCKS_PER_SEC;
	printf("It tooks %.3f seconds\n", tDelta);

	hipFree(dev_buffer);
	return true;
}

void Julia2D::draw()
{
	glBegin(GL_POINTS);
	for (int y = 0; y < height; ++y)
	{
		for (int x = 0; x < width; ++x)
		{
			int i = width * y + x;
			glColor3ub(
				points[i].color[0],
				points[i].color[1],
				points[i].color[2]
			);
			glVertex2i(x, y);
		}
	}
	glEnd();
}