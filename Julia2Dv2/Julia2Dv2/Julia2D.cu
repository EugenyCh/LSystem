#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "Julia2D.cuh"
#include "hipComplex.cuh"
#include <GL/freeglut.h>
#include <GL/freeglut_ext.h>
#include <stdio.h>
#include <time.h>

__global__ void kernel(byte* buffer, const int width, const int height, const float cx, const float cy, const int iters)
{
	int offset = threadIdx.x + blockDim.x * blockIdx.x;
	int x = offset % width;
	int y = offset / width;
	if (y >= height)
		return;

	// Compute point at this position
	int halfWidth = width >> 1;
	int halfHeight = height >> 1;
	int halfSide = MAX(halfWidth, halfHeight);
	float jx = 2.0f * (float)(x - halfWidth) / halfSide;
	float jy = 2.0f * (float)(y - halfHeight) / halfSide;
	hipComplex c(cx, cy);
	hipComplex z(jx, jy);

	// Iterating
	int i;
	for (i = 0; i < iters; ++i)
	{
		z = z * z + c;
		if (z.sqrMagnitude() > 4.0f)
			break;
	}
	float k = (float)i / iters;

	// Setting point color
	buffer[offset * 3] = (byte)(k * 255);
	buffer[offset * 3 + 1] = (byte)(k * 255);
	buffer[offset * 3 + 2] = (byte)(k * 255);
}

Julia2D::Julia2D(float cx, float cy)
{
	this->cx = cx;
	this->cy = cy;
}

bool Julia2D::compute(size_t width, size_t height, int iters)
{
	if (points)
		delete[] points;
	this->width = width;
	this->height = height;
	int side = MAX(width, height);

	const size_t sz = width * height;
	points = new byte[sz * 3];
	byte* dev_buffer;

	if (hipMalloc((void**)&dev_buffer, sz * 3) != hipSuccess)
	{
		printf("Error on creating buffer of pixels in GPU\n");
		return false;
	}

	printf("Rendering %dx%d\n", width, height);
	int threads = 1024;
	int blocks = (sz + threads - 1) / threads;
	clock_t tStart = clock();
	kernel<<<blocks, threads>>>(dev_buffer, width, height, cx, cy, 200);
	hipDeviceSynchronize();
	clock_t tFinish = clock();
	double tDelta = (double)(tFinish - tStart) / CLOCKS_PER_SEC;
	printf("It tooks %.3f seconds\n", tDelta);

	printf("Moving\n");
	if (hipMemcpy((void*)points, dev_buffer, sz * 3, hipMemcpyDeviceToHost) != hipSuccess)
	{
		printf("Error on getting buffer of pixels from GPU\n");
		return false;
	}

	hipFree(dev_buffer);
	return true;
}

void Julia2D::draw()
{
	glBegin(GL_POINTS);
	for (int y = 0; y < height; ++y)
	{
		for (int x = 0; x < width; ++x)
		{
			int i = (width * y + x) * 3;
			glColor3ub(
				points[i],
				points[i + 1],
				points[i + 2]
			);
			glVertex2i(x, y);
		}
	}
	glEnd();
}