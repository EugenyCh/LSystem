#include "hip/hip_complex.h"

__device__ hipComplex::hipComplex(float a, float b) : r(a), i(b) {}

__device__ float hipComplex::sqrMagnitude(void)
{
	return r * r + i * i;
}

__device__ hipComplex hipComplex::operator*(const hipComplex& a)
{
	return hipComplex{ r * a.r - i * a.i, i * a.r + r * a.i };
}

__device__ hipComplex hipComplex::operator+(const hipComplex& a)
{
	return hipComplex{ r + a.r, i + a.i };
}