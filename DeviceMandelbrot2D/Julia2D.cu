#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "Julia2D.cuh"
#include "hipComplex.cuh"
#include <GL/freeglut.h>
#include <GL/freeglut_ext.h>
#include <stdio.h>
#include <time.h>

__global__ void kernel(byte* buffer, const int side, const float cx, const float cy, const int iters)
{
	int offset = threadIdx.x + blockDim.x * blockIdx.x;
	if (offset >= side * side)
		return;
	int x = offset % side;
	int y = offset / side;

	// Compute point at this position
	int halfSide = side >> 1;
	float jx = 2.0f * (float)(x - halfSide) / halfSide;
	float jy = 2.0f * (float)(y - halfSide) / halfSide;
	hipComplex c(cx, cy);
	hipComplex z(jx, jy);

	// Iterating
	int i;
	for (i = 0; i < iters; ++i)
	{
		z = z * z + c;
		if (z.sqrMagnitude() > 4.0f)
			break;
	}
	float k = (float)i / iters;

	// Setting point color
	offset *= 3;
	buffer[offset] = (byte)(k * 255);
	buffer[offset + 1] = (byte)(k * 255);
	buffer[offset + 2] = (byte)(k * 255);
}

Julia2D::Julia2D(float cx, float cy)
{
	this->cx = cx;
	this->cy = cy;
}

bool Julia2D::compute(size_t width, size_t height, int iters)
{
	if (points)
		delete[] points;
	this->width = width;
	this->height = height;
	int side = MAX(width, height);

	const size_t sz = side * side;
	points = new byte[sz * 3];
	byte* dev_buffer;

	if (hipMalloc((void**)&dev_buffer, sz * 3) != hipSuccess)
	{
		printf("Error on creating buffer of pixels in GPU\n");
		return false;
	}

	printf("Rendering %d^2\n", side);
	int threads = 1024;
	int blocks = (sz + threads - 1) / threads;
	clock_t tStart = clock();
	kernel<<<blocks, threads>>>(dev_buffer, side, cx, cy, 200);
	hipDeviceSynchronize();
	clock_t tFinish = clock();
	double tDelta = (double)(tFinish - tStart) / CLOCKS_PER_SEC;
	printf("It tooks %.3f seconds\n", tDelta);

	printf("Moving\n");
	if (hipMemcpy((void*)points, dev_buffer, sz * 3, hipMemcpyDeviceToHost) != hipSuccess)
	{
		printf("Error on getting buffer of pixels from GPU\n");
		return false;
	}

	hipFree(dev_buffer);
	return true;
}

void Julia2D::draw()
{
	glBegin(GL_POINTS);
	int side = MAX(width, height);
	int shiftX = (width - side) / 2 - width / 2;
	int shiftY = (height - side) / 2 - height / 2;
	for (int y = 0; y < side; ++y)
	{
		for (int x = 0; x < side; ++x)
		{
			int i = (side * y + x) * 3;
			glColor3ub(
				points[i],
				points[i + 1],
				points[i + 2]
			);
			glVertex2f(
				shiftX + x,
				shiftY + y
			);
		}
	}
	glEnd();
}